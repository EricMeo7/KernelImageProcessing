#include <iostream>
#include <chrono>
#include <string>
#include "image.h"
#include "kernel.h"
using namespace std;


#define GAUSSIAN        	"gaussian"
#define SHARPENING      	"sharpen"
#define EDGE  				"edge_detect"
#define LAPLACIAN     		"laplacian"
#define GAUSSIAN_LAPLACIAN  "gaussian_laplacian"
#define LAST			"last"

#define CUDA_GLOBAL		"global"
#define CUDA_CONSTANT	"constant"
#define CUDA_SHARED		"shared"

#define OUTPUT_FOLDER   "image_output/"
#define IMAGE_EXT       ".jpg"


int main() 
{
	string filterType[5] = {"GAUSSIAN", "SHARPEN", "EDGE", "LAPLACIAN", "GAUSSIAN_LAPLACIAN"};
	Image img;
	img.loadImage("image_input/image_4k.png");
	CudaMemType cudaType = CudaMemType::SHARED;
	//CudaMemType cudaType = CudaMemType::GLOBAL;
  
	for (int i = 0; i < 5; i++)
	{
 
		Kernel filter = Kernel();
		
		if (filterType[i] == "GAUSSIAN"){
			filter.setGaussianFilter(7, 7, 1);
		} else if (filterType[i] == "SHARPEN"){
			filter.setSharpenFilter();
		} else if (filterType[i] == "EDGE"){
			filter.setEdgeDetectionFilter();
		} else if (filterType[i] == "LAPLACIAN"){
			filter.setLaplacianFilter();
		} else if (filterType[i] == "GAUSSIAN_LAPLACIAN"){
			filter.setGaussianLaplacianFilter();
		} else {
			filter.setGaussianFilter(5, 5, 2);
			
		}
		
		Image newMtImg;
		Image newNpImg;

		// Init the CUDA device
		hipFree(0);	

		
		// Executing image processing
		auto start = std::chrono::high_resolution_clock::now();
		bool cudaResult = img.imageProcessing(newMtImg, filter, cudaType);
		auto end = std::chrono::high_resolution_clock::now();

		std::cout << std::endl;


		// Evaluating execution times and save results
		if (cudaResult) {
			auto multithreadDuration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
			std::cout << "Total CUDA Execution time: " << multithreadDuration << " μs" << std::endl;
			newMtImg.saveImage(std::string(std::string(OUTPUT_FOLDER) + 
								"result_" + filterType[i] +
								std::string(IMAGE_EXT)).c_str());
		}

	}
}
